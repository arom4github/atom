#include "hip/hip_runtime.h"
#include "../../include/maxwell.h"

/* Common Block Declarations */

typedef long int integer;
typedef double doublereal;

struct cag05b_1_ {
    doublereal store1, store2;
};

#define cag05b_1 (*(struct cag05b_1_ *) &cag05b_)

struct cag05a_1_ {
    integer ix, iy, iz;
};

#define cag05a_1 (*(struct cag05a_1_ *) &cag05a_)

/* Initialized data */

struct cag05b {
    doublereal e_1[2];
} cag05b_ = {1., -1.};

struct cag05a {
    integer e_1[3];
} cag05a_ = {1, 255, 25555};


/* Table of constant values */

//static integer c__9 = 9;
//static integer c__1 = 1;
//static integer c__3 = 3;
//static integer c__5 = 5;
//static integer c__0 = 0;
//static integer c__2 = 2;
//static doublereal c_b172 = 1.;
//static integer c__12 = 12;
//static doublereal c_b417 = 1.1424;
//static doublereal c_b419 = .5712;
//static doublereal c_b429 = 1.5;
//static integer c__20000 = 20000;
//static doublereal c_b581 = 0.;
//static doublereal c_b582 = .11200000000000002;
//static doublereal c_b587 = .14;
//static doublereal c_b589 = .8;
//static doublereal c_b614 = .001;

/* ------------------------------------------------------ */
/* ������� �.�., */
doublereal g05dde_(doublereal *a, doublereal *b, int dbg_print) {
    /* Initialized data */

    static doublereal one = 1.;
    static doublereal half = .5;
    static doublereal d__[41] = {0., .674489750196082, 1.150349380376008,
                                 1.534120544352546, 1.862731867421652, 2.153874694061456,
                                 2.417559016236505, 2.66006746861746, 2.885634912426757,
                                 3.097269078198785, 3.297193345691964, 3.487104104114431,
                                 3.668329285121323, 3.841930685501911, 4.008772594168585,
                                 4.169569323349106, 4.324919040826046, 4.475328424654204,
                                 4.621231001499247, 4.763001034267814, 4.900964207963193,
                                 5.035405969463927, 5.166578119728753, 5.294704084854598,
                                 5.419983174916868, 5.54259405780294, 5.662697617459439,
                                 5.780439324478935, 5.895951216739571, 6.009353565530745,
                                 6.120756285971941, 6.230260137989044, 6.33795775455379,
                                 6.443934526538564, 6.548269367831731, 6.651035379893011,
                                 6.752300431407015, 6.852127665896068, 6.95057594791675,
                                 7.047700256664409, 7.14355203435219};

    /* System generated locals */
    doublereal ret_val;

    /* Local variables */
    static integer n;
    static doublereal t, u, v, w, x;
    extern doublereal wrapg05cae_(doublereal *, int);

    u = cag05b_1.store1;
    for (n = 1; n <= 39; ++n) {
        if (u > half) {
            goto L40;
        }
        u += u;
/* L20: */
    }
    n = 40;
    L40:
    t = d__[n - 1];
    u = wrapg05cae_(&x, dbg_print);
    L60:
    w = (d__[n] - t) * u;
    v = w * (w * half + t);
    L80:
    u = wrapg05cae_(&x, dbg_print);
    if (v <= u) {
        goto L100;
    }
    v = wrapg05cae_(&x, dbg_print);
    if (u > v) {
        goto L80;
    }
    u = (v - u) / (one - u);
    goto L60;
    L100:
    u = (u - v) / (one - v);
    if (u > half) {
        goto L120;
    }
    cag05b_1.store1 = u + u;
    ret_val = *a + *b * (w + t);
    return ret_val;
    L120:
    cag05b_1.store1 = u + u - one;
    ret_val = *a - *b * (w + t);
    return ret_val;
} /* g05dde_ */

/* ------------------------------------------------------------------ */
doublereal g05cae_(doublereal *x, int dbg_print) {
    /* System generated locals */
    doublereal ret_val;

    /* Local variables */
    static doublereal ai;
    static integer ii;
    static doublereal ax, ay, az;

    cag05a_1.ix = (cag05a_1.ix - cag05a_1.ix / 177 * 177) * 171 - (
            cag05a_1.ix / 177 << 1);
    cag05a_1.iy = (cag05a_1.iy - cag05a_1.iy / 176 * 176) * 172 - (
            cag05a_1.iy / 176 << 1);
    cag05a_1.iz = (cag05a_1.iz - cag05a_1.iz / 178 * 178) * 170 - (
            cag05a_1.iz / 178 << 1);
    if (cag05a_1.ix < 0) {
        cag05a_1.ix += 30269;
    }
    if (cag05a_1.iy < 0) {
        cag05a_1.iy += 30307;
    }
    if (cag05a_1.iz < 0) {
        cag05a_1.iz += 30323;
    }
    ax = (doublereal) cag05a_1.ix;
    ay = (doublereal) cag05a_1.iy;
    az = (doublereal) cag05a_1.iz;
    ai = ax / 30269. + ay / 30307. + az / 30323.;
    ii = (integer) ai;
    ret_val = ai - ii;
    return ret_val;
} /* g05cae_ */

doublereal wrapg05cae_(doublereal *x, int dbg_print) {
    static int n = 0;
    double t = g05cae_(x, dbg_print);
// #ifdef DEBUG
    n++;
    if (dbg_print == 1) {
        std::cout << n << " " << t << std::endl;
    }
// #endif

    return t;
}

doublereal wrapg05dde_(doublereal *a, doublereal *b, int dbg_print) {
    return g05dde_(a, b, dbg_print);
}

double rnd_uniform(int dbg_print) {
    doublereal x;

    return (double) wrapg05cae_(&x, dbg_print);
}

double rnd_gaussian(double a, double b, int dbg_print) {
    return (double) wrapg05dde_(&a, &b, dbg_print);
}

int in_range(double z0, double z, double z1) {
    return ((z > z0) && (z < z1)) || ((fabs(z - z0) < 1e-13) && (fabs(z - z1) < 1e-13));
}

int AllocateBinaryParticleArraysOneSort(double **x, double **y, double **z, double **px, double **py, double **pz, double **m, int total_particles) {
    *x = new double[total_particles];
    *y = new double[total_particles];
    *z = new double[total_particles];

    *px = new double[total_particles];
    *py = new double[total_particles];
    *pz = new double[total_particles];

    *m = new double[total_particles];

    return 0;
}

void AllocateBinaryParticlesArrays(ParticleArrays *ions, ParticleArrays *electrons, ParticleArrays *beam_electrons) {
    AllocateBinaryParticleArraysOneSort(&(ions->x), &(ions->y), &(ions->z),
                                        &(ions->px), &(ions->py), &(ions->pz),
                                        &(ions->m), ions->total);

    AllocateBinaryParticleArraysOneSort(&(electrons->x), &(electrons->y), &(electrons->z),
                                        &(electrons->px), &(electrons->py), &(electrons->pz),
                                        &(electrons->m), electrons->total);

    AllocateBinaryParticleArraysOneSort(&(beam_electrons->x), &(beam_electrons->y), &(beam_electrons->z),
                                        &(beam_electrons->px), &(beam_electrons->py), &(beam_electrons->pz),
                                        &(beam_electrons->m), beam_electrons->total);
}

int InitUniformMaxwellianParticles(ParticlesConfig *pC, int jmb, int *jmb_real) {
    int beamf = pC->beamPlasma, meh = pC->meh;
    double tex0 = pC->tempX, tey0 = pC->tempY, tez0 = pC->tempZ;
    double beam_lx = pC->beam_lx, beam_ly = pC->beam_ly, beam_lz = pC->beam_lz;
    double lx = pC->lx, ly = pC->ly, lz = pC->lz;
    double Tb = pC->beamVelDisp, rimp = pC->beamImp, rbd = pC->beamPlasmaDensityRat;

    double *xi = pC->ions->x, *yi = pC->ions->y, *zi = pC->ions->z, *ui = pC->ions->px, *vi = pC->ions->py, *wi = pC->ions->pz;
    double *xb = pC->beam->x, *yb = pC->beam->y, *zb = pC->beam->z, *ub = pC->beam->px, *vb = pC->beam->py, *wb = pC->beam->pz;
    double *xf = pC->electrons->x, *yf = pC->electrons->y, *zf = pC->electrons->z, *uf = pC->electrons->px, *vf = pC->electrons->py, *wf = pC->electrons->pz;

    double x, y, z, vb0, d__1, d__2, d__3, vy, vz, termx, gb0;
    double vf01, vf02, pinv1, pinv2, mfrq = 0.0;
    double *ux, *uy, *uz;
    double beam_y_max, beam_y_min, beam_sh;
    double beam_z_max, beam_z_min, beam_shz;

    beam_sh = (ly - beam_ly) / 2;
    beam_y_max = ly - beam_sh;
    beam_y_min = beam_sh;

    beam_shz = (lz - beam_lz) / 2;
    beam_z_max = lz - beam_shz;
    beam_z_min = beam_shz;

    int j;

    ux = new double[jmb];
    uy = new double[jmb];
    uz = new double[jmb];

    for (j = 1; j <= jmb; j++) {
        z = lz * rnd_uniform(0);
        y = meh * ly + ly * rnd_uniform(0);
        x = lx * rnd_uniform(0);

        xi[j - 1] = x;
        yi[j - 1] = y;
        zi[j - 1] = z;
        ui[j - 1] = 0.0;
        vi[j - 1] = 0.0;
        wi[j - 1] = 0.0;
    }

/* ****************** BEAM **************************************** */
    *jmb_real = 0;
    for (j = 1; j <= jmb; j++) {
        y = yi[j - 1];
        z = zi[j - 1];
        if ((xi[j - 1] < beam_lx) && (y < beam_y_max) && (y > beam_y_min) && in_range(beam_z_min, z, beam_z_max)) {
            xb[*jmb_real] = xi[j - 1];
            yb[*jmb_real] = yi[j - 1];
            zb[*jmb_real] = zi[j - 1];
            vb0 = rnd_gaussian(0.0, Tb * rimp, 0);
            ux[*jmb_real] = vb0 + rimp;
            uy[*jmb_real] = rnd_gaussian(0.0, Tb * rimp, 0);
            uz[*jmb_real] = rnd_gaussian(0.0, Tb * rimp, 0);
#ifdef DEBUG_INITIAL_PARTICLE_PRINTS
            printf("ion %10d %25.15e %25.15e %25.15e \n",j,xi[j - 1],yi[j - 1],zi[j - 1]);
#endif
            (*jmb_real)++;
        }
    }

    //1st beam particle impulse:    0.20296063288436139
    for (j = 1; j <= *jmb_real; j++) {
        d__1 = ux[j - 1];
        d__2 = uy[j - 1];
        d__3 = uz[j - 1];

        vb0 = sqrt(1.0 - d__1 * d__1 - d__2 * d__2 - d__3 * d__3);

        ub[j - 1] = ux[j - 1] / vb0;

        double t = fabs(ub[j - 1] - 0.20296063288436139);

        vb[j - 1] = uy[j - 1] / vb0;
        wb[j - 1] = uz[j - 1] / vb0;
#ifdef DEBUG_INITIAL_PARTICLE_PRINTS
        printf("beam %10d %25.15e  %25.15e    %25.15e %25.15e %25.15e   %25.15e \n", j,  xb[j - 1],yb[j - 1],vb0,    ub[j-1],vb[j - 1],wb[j - 1]);
#endif
    }

    for (j = 1; j <= jmb; j++) {
        xf[2 * j - 1 - 1] = xi[j - 1];
        yf[2 * j - 1 - 1] = yi[j - 1];
        zf[2 * j - 1 - 1] = zi[j - 1];

        xf[2 * j - 1] = xi[j - 1];
        yf[2 * j - 1] = yi[j - 1];
        zf[2 * j - 1] = zi[j - 1];

//      FIRST SETTING TRANVERSE
//      razbros v skorostyax

        vy = rnd_gaussian(0.0, tey0, 0);
        vz = rnd_gaussian(0.0, tez0, 0);

//      INVERSE CURRENT

        termx = rnd_gaussian(0.0, tex0, 0);

        gb0 = pow(1.0 + pow(ub[j - 1], 2) + pow(vb[j - 1], 2) + pow(wb[j - 1], 2), -0.5);

        vb0 = ub[j - 1] * gb0;
        if ((beamf == 1) && ((xi[j - 1] < beam_lx) && (yi[j - 1] < beam_y_max) && (yi[j - 1] > beam_y_min))) {
            vf01 = -rbd * vb0 + termx;
            vf02 = -rbd * vb0 - termx;
        } else {
            vf01 = +termx;
            vf02 = -termx;
        }

        pinv1 = vf01 * pow((1.0 - pow(vf01, 2) - vy * vy - vz * vz), -0.5);
        pinv2 = vf02 * pow((1.0 - pow(vf02, 2) - vy * vy - vz * vz), -0.5);

        vf[2 * j - 2] = vy * pow((1.0 - pow(vf01, 2) - vy * vy - vz * vz), -0.5);
        vf[2 * j - 1] = -vy * pow((1.0 - pow(vf02, 2) - vy * vy - vz * vz), -0.5);

        wf[2 * j - 2] = vz * pow((1.0 - pow(vf01, 2) - vy * vy - vz * vz), -0.5);
        wf[2 * j - 1] = -vz * pow((1.0 - pow(vf02, 2) - vy * vy - vz * vz), -0.5);

        uf[2 * j - 2] = pinv1 + 0.01 * sin(mfrq * 2.0 * M_PI * xf[2 * j - 2] / lx);
        uf[2 * j - 1] = pinv2 + 0.01 * sin(mfrq * 2.0 * M_PI * xf[2 * j - 1] / lx);

#ifdef DEBUG_INITIAL_PARTICLE_PRINTS
        printf("electron %10d %25.15e %25.15e %25.15e %25.15e \n",2*j-2,yi[j - 1],uf[2*j-2],vf[2*j-2],wf[2*j-2]);
        printf("electron %10d %25.15e %25.15e %25.15e %25.15e \n",2*j-1,yi[j - 1],uf[2*j-1],vf[2*j-1],wf[2*j-1]);
#endif
    }

    delete[] ux;
    delete[] uy;
    delete[] uz;

    return 0;
} /* start_ */

int getMassCharge(ParticleArrays *ions, ParticleArrays *electrons, ParticleArrays *beam_electrons, double ni, double rbd, int lp) {
    ions->m[0] = (ni + rbd) / lp;
    electrons->m[0] = -ni / lp / 2.0;     //!!!!!!
    beam_electrons->m[0] = -rbd / lp;

    ions->q_m = 1.0 / 1836.0;
    electrons->q_m = -1.0;
    beam_electrons->q_m = -1.0;

    return 0;
}

int AllocateMemoryForArrays(int total, ParticleArrays *ions, ParticleArrays *electrons, ParticleArrays *beam_electrons) {
    ions->total = total;
    electrons->total = 2 * total;
    beam_electrons->total = total;

    AllocateBinaryParticlesArrays(ions, electrons, beam_electrons);

    return 0;
}

int convertParticleArraysToSTLvector(ParticleArrays *pA, particle_sorts sort, std::vector <Particle> &vp) {
    double x, y, z, px, py, pz;

    for (int i = 0; i < pA->total; i++) {
        x = pA->x[i];
        y = pA->y[i];
        z = pA->z[i];
        px = pA->px[i];
        py = pA->py[i];
        pz = pA->pz[i];

        Particle p(x, y, z, px, py, pz, pA->m[0], pA->q_m);

        p.fortran_number = i + 1;
        p.sort = sort;

        vp.push_back(p);
    }

    return 0;
}

int getUniformMaxwellianParticles(std::vector <Particle> &ion_vp, std::vector <Particle> &el_vp, std::vector <Particle> &beam_vp, ParticlesConfig *pC) {
    int total = pC->nx * pC->ny * pC->nz * pC->lp;
    int jmb;

    AllocateMemoryForArrays(total, pC->ions, pC->electrons, pC->beam);

    getMassCharge(pC->ions, pC->electrons, pC->beam, pC->plsmDensity, pC->beamPlasmaDensityRat, pC->lp);

    InitUniformMaxwellianParticles(pC, total, &jmb);

    convertParticleArraysToSTLvector(pC->beam, BEAM_ELECTRON, beam_vp);
    convertParticleArraysToSTLvector(pC->ions, ION, ion_vp);
    convertParticleArraysToSTLvector(pC->electrons, PLASMA_ELECTRON, el_vp);

    return 0;
}
